#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cpu_bitmap.h"

#include <stdio.h>
#include <stdlib.h>

#define DIM 1024

__global__ void kernel(unsigned char *img) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int pixel = x + y * blockDim.x * gridDim.x;

	img[pixel * 4 + 0] = 255 * x / (blockDim.x * gridDim.x);
	img[pixel * 4 + 1] = 255 * y / (blockDim.y * gridDim.y);
	img[pixel * 4 + 2] = 2 * blockIdx.x + 2 * blockIdx.y;
	img[pixel * 4 + 3] = 255;
}

int main(){/*
	// declaracion del bitmap
	CPUBitmap bitmap(DIM, DIM);
	// tama�o en bytes
	size_t size = bitmap.image_size();
	// reserva en el host
	unsigned char *host_bitmap = bitmap.get_ptr();
	// reserva en el device
	unsigned char *dev_bitmap;
	hipMalloc((void**)&dev_bitmap, size);
	
	// generamos el bitmap
	dim3 Nbloques(DIM / 16, DIM / 16);
	dim3 hilosB(16, 16);
	
	kernel <<<Nbloques, hilosB >>> (dev_bitmap);
	// recogemos el bitmap desde la GPU para visualizarlo
	hipMemcpy(host_bitmap, dev_bitmap, size, hipMemcpyDeviceToHost);
	// liberacion de recursos
	hipFree(dev_bitmap);
	// visualizacion y salida
	printf("\n...pulsa ESC para finalizar...");
	bitmap.display_and_exit();
	*/
	return 0;

}
