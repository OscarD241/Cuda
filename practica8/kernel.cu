
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>


#define N 3

__constant__ float dev_A[N][N];

__global__ void cuadrada(float *dev_B){
	// kernel lanzado con un solo bloque y NxN hilos
	int columna = threadIdx.x;
	int fila = threadIdx.y;
	int pos = fila + N * columna;
	dev_B[pos] = 0;
	// cada hilo coloca un elemento de la matriz final
	for(int k = 0 ; k < N ; k++)
		dev_B[pos] += dev_A[columna][k] * dev_A[k][fila];
}


int main(int argc, char** argv){
	float *hst_A, *hst_B;;
	float *dev_B;

	// reserva en el host
	hst_A = (float*)malloc(N*N * sizeof(float));
	hst_B = (float*)malloc(N*N * sizeof(float));
	// reserva en el device
	hipMalloc((void**)&dev_B, N*N * sizeof(float));
	// inicializacion
	for (int i = 0; i < N*N; i++){
		hst_A[i] = (float)i;
	}
	// copia de datos
	hipMemcpyToSymbol(HIP_SYMBOL(dev_A), hst_A, N*N * sizeof(float));
	// dimensiones del kernel
	dim3 Nbloques(1);
	dim3 hilosB(N, N);

	// llamada al kernel bidimensional de NxN hilos
	cuadrada <<<Nbloques, hilosB >>> (dev_B);

	// recogida de datos
	hipMemcpy(hst_B, dev_B, N*N * sizeof(float), hipMemcpyDeviceToHost);
	// impresion de resultados
	printf("Resultado:\n");
	printf("ORIGINAL:\n");
	for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++){
			printf("%2.0f ", hst_A[j + i * N]);
		}
		printf("\n");
	}
	printf("CUADRADA:\n");
	for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++){
			printf("%2.0f ", hst_B[j + i * N]);
		}
		printf("\n");
	}
	// salida
	return 0;
}