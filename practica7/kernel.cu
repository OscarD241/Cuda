
#include "hip/hip_runtime.h"

#include "hip/hip_fp16.h"

#include <stdio.h>
#include <stdlib.h>

#include <iostream>

#define N 16

using namespace std;

__global__ void pi(float *vector, float *res) {
	__shared__ float tmp[N];
	int id = threadIdx.x;
	tmp[id] = 1 / pow(vector[id],2);
	__syncthreads();

	int step = N / 2;

	while (step){
		if (id < step)
			tmp[id] += tmp[id + step];
		__syncthreads();
		step /= 2;
	}

	if (id == 0)
		*res = tmp[id];//sqrt(6 * tmp[id]);
}

int main(){
	hipError_t cudaStatus;
	//host variables
	float *hVec = (float *)malloc(N * sizeof(float));
	float *hSum = (float *)malloc(sizeof(float));
	//device variables
	float *dVec;
	float *dSum;

	cudaStatus = hipMalloc((void **)&dVec, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		cout << "hipMalloc 1 has failed... " << endl;
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dSum, sizeof(float));
	if (cudaStatus != hipSuccess) {
		cout << "hipMalloc 2 has failed... " << endl;
		goto Error;
	}

	for (char i = 0; i < N; i++)
		hVec[i] = (float)i;

	cudaStatus = hipMemcpy(dVec, hVec, N * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		cout << "cudaMemcpy1 has failed... " << endl;
		goto Error;
	}

	pi <<<1, N >>> (dVec,dSum);
	hipDeviceSynchronize();

	cudaStatus = hipMemcpy(hSum, dSum, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		cout << "cudaMemcpy2 has failed... " << endl;
		goto Error;
	}

	cout << "Result = " << *hSum << endl;

Error:
	hipFree(dVec);
	hipFree(dSum);
	free(hVec);
	free(hSum);
	return 0;
}
