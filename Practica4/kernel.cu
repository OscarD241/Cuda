
#include "hip/hip_runtime.h"

#include "hip/hip_fp16.h"

#include <stdio.h>
#include <stdlib.h>
#include <iostream>

using namespace std;

__global__ void kernel(double *vals, double *x) {
	int id = threadIdx.x;
	double a = vals[0];
	double b = vals[1];
	double c = vals[2];
	if (id < 2)
		x[id] = ((-1.0)*(b / (2.0 * a))) + (pow(-1.0, (double)(id + 1)) *
			(sqrt(pow(b, (double) 2.0) - (4.0 * a * c)) / (2.0 * a)));
}

int main(){
    hipError_t cudaStatus;
	//Host variables
	double hstVals[] = { 1.0, 2.0, 0.0 };
	double *hstX;
	//device variables
	double *devVals;
	double *devX;

	hstX = (double *)malloc(2 * sizeof(double));
	cudaStatus = hipMalloc((void **)&devX, 2 * sizeof(double));
	if (cudaStatus != hipSuccess) {
		cout << "hipMalloc has failed... " << endl;
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&devVals, 3 * sizeof(double));
	if (cudaStatus != hipSuccess) {
		cout << "hipMalloc has failed... " << endl;
		goto Error;
	}

	cudaStatus = hipMemcpy(devVals, hstVals, 3 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		cout << "cudaMemcpy1 has failed... " << endl;
		goto Error;
	}

	kernel <<<1, 2 >>> (devVals,devX);
	hipDeviceSynchronize();

	cudaStatus = hipMemcpy(hstX, devX, 2 * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		cout << "cudaMemcpy4 has failed... " << endl;
		goto Error;
	}

	cout << "Results:\n x1 = " << hstX[0] << "\n x2 = " << hstX[1] << endl;
	cout << "Everythyng was ok... :)" << endl;

Error:
	hipFree(devVals);
	hipFree(devX);
    return 0;
}
