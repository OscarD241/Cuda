
#include "hip/hip_runtime.h"


#include <stdio.h>

/*
* Práctica 1
* Objetivo: Obtener una breve descripción de los recursos con los que cuenta el sistema
*/

int main(){
	int counter;											//Contador de dispositivos
	hipDeviceProp_t prop;									//Estructura en la que se almacenarán las características de la GPU

	hipError_t error = hipGetDeviceCount(&counter);		//obtenemos la cantidad de dispositivos que se tiene
	if (error != hipSuccess)
		printf("%s\n", hipGetErrorString(error));
	else {
		printf("Total de dispositivos: %d\n", counter);
		for (int i = 0; i < counter; i++) {
			hipGetDeviceProperties(&prop, i);
			printf("Nombre: %s\n", prop.name);
			printf("Capacidad de cómputo mayor: %d\n",prop.major);
			printf("Capacidad de cómputo menor: %d\n", prop.minor);
			printf("Número de Streaming Multiprocessors: %d\n",prop.multiProcessorCount);
			printf("Máximo número de hilos por bloque: %d\n",prop.maxThreadsPerBlock);
			printf("Máximo número de bloques por grid: %d\n",prop.maxGridSize[0]);
		}
	}

	printf("\n Presiona una tecla para continuar...");
	fflush(stdin);
	char t = getchar();
    return 0;
}

