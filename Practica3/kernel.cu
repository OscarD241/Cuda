
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>

#define N 8

using namespace std;


int main(){
    
    hipError_t cudaStatus;
	float *hstMat;
	float *hstMat2;
	float *devMat;
	float *devMat2;
	unsigned char flg = 0;

	size_t freeMem;
	size_t totMem;

	hstMat = (float *)malloc(N * N * sizeof(float));
	hstMat2 = (float *)malloc(N * N * sizeof(float));
	cudaStatus = hipMalloc((void **)&devMat, N * N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		cout << "hipMalloc has failed... \n" << endl;
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&devMat2, N * N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		cout << "hipMalloc has failed... \n" << endl;
		goto Error;
	}

	srand((int)time(NULL));

	for (unsigned char i = 0; i < N * N; i++)
		hstMat[i] = (float)(rand() % 10);

	cudaStatus = hipMemcpy(devMat,hstMat,N*N*sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		cout << "hipMemcpyHostToDevice has failed... \n";
		goto Error;
	}

	cudaStatus = hipMemcpy(devMat2, devMat, N*N * sizeof(float), hipMemcpyDeviceToDevice);
	if (cudaStatus != hipSuccess) {
		cout << "hipMemcpyDeviceToDevice has failed... \n";
		goto Error;
	}

	cudaStatus = hipMemcpy(hstMat2, devMat2, N*N * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		cout << "hipMemcpyDeviceToHost has failed... \n";
		goto Error;
	}

	for (unsigned char i = 0; i < N * N; i++)
		if (hstMat[i] != hstMat2[i])
			flg++;

	cudaStatus = hipMemGetInfo(&freeMem,&totMem);
	if (cudaStatus != hipSuccess) {
		cout << "hipMemGetInfo has failed... \n";
		goto Error;
	}

	cout << "Free memory: " << freeMem << endl;
	cout << "Total memory: " << totMem << endl;
	cout << "Everything was ok... :)" << endl;
	cout << "total errors: " << (int) flg << endl;

Error:
	free(hstMat);
	free(hstMat2);
	hipFree(devMat);
	hipFree(devMat2);

    return 0;
}

