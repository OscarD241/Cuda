
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#include <iostream>

#define N 16 //vector size
#define BLOCK 5 //block size

using namespace std;

__global__ void sum(float *a, float *b, float *c) {
	int dataId = threadIdx.x + blockDim.x * blockIdx.x;
	if (N > dataId)
		c[dataId] = a[dataId] + b[dataId];
}

int main(){
	hipError_t cudaStatus;
	float *h_vector1 = (float *) malloc(N * sizeof(float));
	float *h_vector2 = (float *)malloc(N * sizeof(float));
	float *h_res = (float *)malloc(N * sizeof(float));
	float *d_vector1;
	float *d_vector2;
	float *d_res;
	int nBloques = 0;
	int i;

	cudaStatus = hipMalloc((void **)&d_vector1, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		cout << "hipMalloc on d_vector1 has failed " << endl;
		goto Error;
	}
	
	cudaStatus = hipMalloc((void **)&d_vector2, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		cout << "hipMalloc on d_vector2 has failed " << endl;
		goto Error;
	}
	
	cudaStatus = hipMalloc((void **)&d_res, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		cout << "hipMalloc on d_res has failed " << endl;
		goto Error;
	}
	
	for ( i = 0; i < N; i++) {
		h_vector1[i] = (float)i;// rand() / RAND_MAX;
		h_vector2[i] = (float) i;// rand() / RAND_MAX;
	}
	
	cudaStatus = hipMemcpy(d_vector1, h_vector1, N * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		cout << "hipMemcpy h_vector1 -> d_vector1 has failed" << endl;
		goto Error;
	}
	
	cudaStatus = hipMemcpy(d_vector2, h_vector2, N * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		cout << "hipMemcpy h_vector2 -> d_vector2 has failed" << endl;
		goto Error;
	}
	
	nBloques = (int) N / BLOCK;
	if (N % BLOCK == 0)
		nBloques++;
	
	sum <<<nBloques, BLOCK >>> (d_vector1, d_vector2, d_res);
	hipDeviceSynchronize();
	
	cudaStatus = hipMemcpy(h_res, d_res, N * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		cout << "hipMemcpy d_res -> h_res has failed" << endl;
		goto Error;
	}
	
	cout << "Vector 1: " << endl;
	for (i = 0; i < N; i++)
		printf("%.2f ", h_vector1[i]);
	
	cout << "\nVector 2: " << endl;
	for (i = 0; i < N; i++)
		printf("%.2f ", h_vector2[i]);

	cout << "\nResult: " << endl;
	for (i = 0; i < N; i++)
		printf("%.2f ",h_res[i]);

		
Error:
	hipFree(d_vector1);
	hipFree(d_vector2);
	hipFree(d_res);
	free(h_vector1);
	free(h_vector2);
	free(h_res);
    return 0;
}

