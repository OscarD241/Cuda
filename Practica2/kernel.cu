
#include "hip/hip_runtime.h"


#include <stdio.h>

/*
 * Práctica 2
 * 
 * Objetivo: Conocer el funcionamiento de las sumas atómicas y el llamado a los kernels de la GPU
 */

__device__ int b = 0;

__global__ void myKernel() {
	int a = 1;
	atomicAdd(&b,a);
}

int main(){
	myKernel <<<1, 12 >>> ();
	hipDeviceSynchronize();
	printf("%d",b);

    return 0;
}



